#include "hip/hip_runtime.h"
/**
 * SOLVER CONFIGUATION (./cuda/src/config.cpp)
 * Global configuration of the solver including the function and intial value are specfied.
 */
#include <math.h>       /* exp */
long int _GLB_N_ = 1024 * 1;  // Probelm Size - Note keep at base 2
long int _GLB_ITR_ = 10;     // Max Solver Iterations
long int _GLB_ITR_LINE_ = 2; // Max Line search iterationsc
double _GLB_EPS_ = 1e-6;      // Value of epsilon, Note this is equal to the tolerence (both residual and linesearch)


using namespace std;

__device__ void _FUNCTION(long int N, double* x , double* rtrn ) {
	for (int i = 0; i <  N - 1; ++i) {
		rtrn[0] += 100 * (x[i + 1] - x[i] * x[i]) * (x[i + 1] - x[i] * x[i]) + (1 - x[i]) * (1 - x[i]);
	};
};

__device__ void FUNCTION(long int N, double* x , double* rtrn ) {
	for (int i = 0; i <  N ; ++i) {
		rtrn[0] += (x[i]-1.0) / (exp ((x[i]-1.0)/.1)-1.0);
	};
};

//\frac{\left(x-1\right)}{e^{\frac{\left(x-1\right)}{.1}}-1}

inline void GUESS(long int  N, vector<double>& rtrn) {
	vector<double>x0 (N, -7);
	rtrn = x0 ;
};
