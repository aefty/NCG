#include "hip/hip_runtime.h"
namespace cuda {

   //
   __global__ void initSpace( long int N ,  double* x, double EPS, double* space) {
      int i = blockDim.x * blockIdx.x + threadIdx.x;
      int j = blockDim.y * blockIdx.y + threadIdx.y;
      space[j * N + i] = x[i] + EPS;
   };

   __global__ void async_grad( long int N , int i,  double EPS, double* x, double* grad) {
      double val = 0.0;

      x[i] -= EPS;
      FUNCTION(N, &x[0], &val);

      x[i] += 2.0 * EPS;
      val = val * -1.0;

      FUNCTION(N, &x[0], &val);
      grad[0] = val / (2.0 * EPS);
   };



   inline void linalg_grad(long int N ,  double EPS, vector<double>& x,  vector<double>& grad) {


      const int blockSize = 1, nStreams = N;
      const int n = N;
      const int streamSize = n / nStreams;
      const int streamBytes = streamSize * sizeof(double);
      const int bytes = n * sizeof(double);

      // allocate pinned host memory and device memory
      double* x, *_x;
      CUDA_ERR_CHECK( hipHostMalloc((void**)&x, bytes) );      // host pinned
      CUDA_ERR_CHECK( hipMalloc((void**)&_x, bytes) ); // device

      // create events and streams
      hipEvent_t startEvent, stopEvent, dummyEvent;
      hipStream_t stream[nStreams];
      CUDA_ERR_CHECK( hipEventCreate(&startEvent) );
      CUDA_ERR_CHECK( hipEventCreate(&stopEvent) );
      CUDA_ERR_CHECK( hipEventCreate(&dummyEvent) );


      // asynchronous version 1: loop over {copy, kernel, copy}
      //memset(a, 0, bytes);
      CUDA_ERR_CHECK( hipEventRecord(startEvent, 0) );

      for (int i = 0; i < nStreams; ++i) {

         CUDA_ERR_CHECK( hipStreamCreate(&stream[i]) );



         int offset = i * streamSize;
         CUDA_ERR_CHECK( hipMemcpyAsync(&d_a[offset], &a[offset],
                                         streamBytes, hipMemcpyHostToDevice,
                                         stream[i]) );
         kernel <<< streamSize / blockSize, blockSize, 0, stream[i] >>> (d_a, offset);
         CUDA_ERR_CHECK( hipMemcpyAsync(&a[offset], &d_a[offset],
                                         streamBytes, hipMemcpyDeviceToHost,
                                         stream[i]) );
      }

      CUDA_ERR_CHECK( hipEventRecord(stopEvent, 0) );
      CUDA_ERR_CHECK( hipEventSynchronize(stopEvent) );

      // cleanup
      checkCuda( hipEventDestroy(startEvent) );
      checkCuda( hipEventDestroy(stopEvent) );
      checkCuda( hipEventDestroy(dummyEvent) );

      for (int i = 0; i < nStreams; ++i) {
         checkCuda( hipStreamDestroy(stream[i]) );
      }

      hipFree(_x);
      hipHostFree(x);

   };
   /*
   inline void linalg_grad(long int N ,  double EPS, vector<double>& x,  vector<double>& grad) {

      // Create streams
      hipStream_t stream[N];
      int streamLength = 1;
      int streamSize = streamLength * sizeof(double);
      size_t s = x.size() * sizeof(double);

      for (int i = 0; i < N; i++) {

         double* _x;
         CUDA_ERR_CHECK(hipMalloc(_x, s));

         double* _gradi;
         CUDA_ERR_CHECK(hipMalloc(_gradi, sizeof(double)));

         hipEvent_t startEvent, stopEvent;

         CUDA_ERR_CHECK(hipStreamCreate(&stream[i]));
         //CUDA_ERR_CHECK( hipEventRecord(startEvent, stream[i]) );

         CUDA_ERR_CHECK(hipMemcpyAsync( _x, &x[0], streamSize, hipMemcpyHostToDevice, stream[i])) ;

         async_grad <<< 1, 1, 0, stream[i]>>> (N, i, EPS, _x, _gradi);

         CUDA_ERR_CHECK(hipMemcpyAsync( &grad[i], _gradi,  streamSize, hipMemcpyDeviceToHost, stream[i])) ;

       //  CUDA_ERR_CHECK( hipEventRecord(stopEvent, stream[i]) );
       //  CUDA_ERR_CHECK( hipEventSynchronize(stopEvent) )
         hipFree(_x);
         hipFree(_gradi);
      }
   };
   */
};

