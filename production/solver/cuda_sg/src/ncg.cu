#include "hip/hip_runtime.h"
/**
 * NON LINEAR CG SOLVER (./cuda_sg/src/ncg.cu)
 * Main solver file.
 */

#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <iostream>
#include <vector>
#include <time.h>
#include <algorithm>

#include "config.cu"
#include "lib/util.cpp"
#include "lib/math.cpp"
#include "lib/lineSearch.cu"
#include "lib/json.cpp"

using namespace std;

int main(int argc, char* argv[]) {

	int showX = 0;

	if (argc > 1) { _GLB_N_ = (long int) _GLB_N_ * atof(argv[1]); }

	if (argc > 2) { showX = atoi(argv[2]); }

	gpu::deviceSpecs();

	/**
	 * CODE BLOCK 1
	 * Initilization
	 */

	JSON json;

	vector<double> x0; GUESS(_GLB_N_, x0); double* _x0 = (double*) gpu::alloc(x0);
	vector<double> x1(_GLB_N_);
	vector<double> p(_GLB_N_); double* _p = (double*) gpu::alloc(p);
	vector<double> vtemp(_GLB_N_);
	vector<double> g00(_GLB_N_);
	vector<double> g01(_GLB_N_);
	vector<double> g1(_GLB_N_);

	double gg0 = 0.0;
	double gg1 = 0.0;
	double B = 0.0;

	double tol = _GLB_EPS_ + 1.0;
	int itr = 0;
	double  min_i = 0;
	double alpha = 1;
	double h = _GLB_EPS_;

	std::vector<double> alhpa_history(100, 0);
	std::vector<double> m_history(100, 0);

	// ~50% staturated
	long int range = 128;

	vector<double> space(range * _GLB_N_, 0.0); double* _space = (double*) gpu::alloc(space);
	vector<double> func_val(range, 0.0); double* _func_val = (double*) gpu::alloc(func_val);

	double t_lineSearch = 0.0;
	clock_t t_start = clock();

	// BEGIN NCG
	{
		cpu::linalg_grad(_GLB_N_, _GLB_EPS_, x0, p);
		cpu::linalg_sdot( -1.0, p, p);



		cpu::linalg_dot(p, p, gg0);

		for (int i = 0; i < p.size(); ++i) {
			cout << " " << p[i];
		}

		cout << endl;
		x1 = x0;

		while (tol > _GLB_EPS_ && itr < _GLB_ITR_) {

			cout << "| Tol :" << tol << endl;
			clock_t t_lineSearch_start = clock();

			/**
			* CODE BLOCK 2
			* Line Search
			*/
			{
				gpu::alloc(x0, _x0);
				gpu::alloc(p, _p);

				//h = 1 / gg0;

				gpu::spcl <<< 128 , _GLB_N_ * range / 128 + 1 >>>   (_GLB_N_, range, _x0 , _p, h , _space);
				gpu::fv <<< 128 , range + 1 >>> (_GLB_N_, range, _space ,  _func_val);

				CUDA_ERR_CHECK(hipDeviceSynchronize());
				gpu::unalloc(_func_val, func_val );

				min_i = distance(func_val.begin(), min_element(func_val.begin(), func_val.end()));

				alpha = (min_i - range / 2)  * h;

				m_history[itr] = (min_i - range / 2);
				alhpa_history[itr] = alpha;


			}
			// END LINE SEARCH

			/**
			* CODE BLOCK 3
			* Direction
			*/
			cpu::linalg_add (1.0, x0, alpha, p, x1);

			t_lineSearch += (clock() - t_lineSearch_start) / (double) CLOCKS_PER_SEC;

			cpu::linalg_grad(_GLB_N_, _GLB_EPS_, x1, g1);
			cpu::linalg_dot(g1, g1, gg1);
			B = gg1 / gg0;

			//% p = -g1 + B * p;
			cpu::linalg_add(-1.0, g1, B, p, p);

			//% tol = norm(x1 - x0)
			cpu::linalg_add(1.0, x1, -1.0, x0, vtemp);
			cpu::linalg_dot(vtemp, vtemp, tol);
			tol = pow(tol , 0.5) / _GLB_N_;
			gg0 = gg1;

			x0 = x1;
			itr ++;
		}
	}
	//END NCG


	// Get timining and metrics
	double t_run = (clock() - t_start) / (double) CLOCKS_PER_SEC;
	double rate = (double)_GLB_N_ / t_run;
	t_lineSearch = t_lineSearch;

	gpu::unalloc(_space, space);
	gpu::unalloc(_space);

	double x_max = *max_element(std::begin(x1), std::end(x1));
	double x_min = *min_element(std::begin(x1), std::end(x1));

	// Output
	json.append("size", _GLB_N_);
	json.append("itr", itr);
	json.append("conv", tol);
	json.append("run_time", t_run);
	json.append("line_search_time", t_lineSearch);
	json.append("rate", rate);
	json.append("x_max", x_max);
	json.append("x_min", x_min);
	json.append("alpha", alhpa_history);
	json.append("m_history", m_history);

	json.append("space", space);
	json.append("func_val", func_val);
	json.append("p", p);

	if (showX) {
		json.append("x", x1);
	}

	cout << "\n\n";
	cout << json.dump();
	cout << "\n\n";
	return 0;
}