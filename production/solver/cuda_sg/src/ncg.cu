#include "hip/hip_runtime.h"
/**
 * NON LINEAR CG SOLVER (./cuda_sg/src/ncg.cu)
 * Main solver file.
 */

#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <iostream>
#include <vector>
#include <time.h>
#include <algorithm>

#include "config.cu"
#include "lib/util.cpp"
#include "lib/math.cpp"
#include "lib/lineSearch.cu"
#include "lib/json.cpp"

using namespace std;

int main(int argc, char* argv[]) {

	int showX = 0;

	if (argc > 1) { _GLB_N_ = (long int) _GLB_N_ * atof(argv[1]); }

	if (argc > 2) { showX = atoi(argv[2]); }

	gpu::deviceSpecs();

	/**
	 * CODE BLOCK 1
	 * Initilization
	 */

	JSON json;

	vector<double> x0; GUESS(_GLB_N_, x0); double* _x0 = (double*) gpu::alloc(x0);
	vector<double> x1(_GLB_N_);
	vector<double> p(_GLB_N_); double* _p = (double*) gpu::alloc(p);
	vector<double> vtemp(_GLB_N_);
	vector<double> g00(_GLB_N_);
	vector<double> g01(_GLB_N_);
	vector<double> g1(_GLB_N_);

	double gg0 = 0.0;
	double gg1 = 0.0;
	double B = 0.0;

	double tol = _GLB_EPS_ + 1.0;
	int itr = 0;
	double  min_i = 0;
	double alpha = 1;
	double h = _GLB_EPS_;

	// ~50% staturated
	int TPB_2D = 16 ;
	long int range = 16;

	int block_x = (_GLB_N_ / TPB_2D) < 1 ? 1 : (_GLB_N_ / TPB_2D) ;
	int block_y = range < 1 ? 1 : range ;

	dim3 nm_tpb (128);
	dim3 nm_blocks(_GLB_N_*range/128);

	dim3 ln_tpb (128);
	dim3 ln_blocks(_GLB_N_ / 128) ;

	vector<double> space(range * _GLB_N_, 0.0); double* _space = (double*) gpu::alloc(space);
	vector<double> func_val(range, 0.0); double* _func_val = (double*) gpu::alloc(func_val);

	double t_lineSearch = 0.0;
	clock_t t_start = clock();

	// BEGIN NCG
	{
		cpu::linalg_grad(_GLB_N_, _GLB_EPS_, x0, p);
		cpu::linalg_sdot( -1.0, p, p);

		cpu::linalg_dot(p, p, gg0);

		x1 = x0;

		while (tol > _GLB_EPS_ && itr < _GLB_ITR_) {

			cout << "| Tol :" << tol << endl;
			clock_t t_lineSearch_start = clock();

			/**
			* CODE BLOCK 2
			* Line Search
			*/
			{
				gpu::alloc(x0, _x0);
				gpu::alloc(p, _p);

				gpu::spcl <<< nm_tpb , nm_blocks>>>   (_GLB_N_, range, _x0 , _p, h , _space);
			//	gpu::fv <<<ln_tpb , ln_blocks>>> (_GLB_N_, range, _space ,  _func_val);

				CUDA_ERR_CHECK(hipDeviceSynchronize());
				gpu::unalloc(_func_val, func_val );

				for (int i = 0; i < func_val.size(); i++) {
					if (func_val[i] < func_val[min_i]) {
						min_i = i;
					}
				}

				alpha = min_i * h;
			}
			// END LINE SEARCH
goto end;

			/**
			* CODE BLOCK 3
			* Direction
			*/
			cpu::linalg_add (1.0, x0, alpha, p, x1);

			t_lineSearch += (clock() - t_lineSearch_start) / (double) CLOCKS_PER_SEC;

			cpu::linalg_grad(_GLB_N_, _GLB_EPS_, x1, g1);
			cpu::linalg_dot(g1, g1, gg1);
			B = gg1 / gg0;

			//% p = -g1 + B * p;
			cpu::linalg_add(-1.0, g1, B, p, p);

			//% tol = norm(x1 - x0)
			cpu::linalg_add(1.0, x1, -1.0, x0, vtemp);
			cpu::linalg_dot(vtemp, vtemp, tol);
			tol = pow(tol , 0.5) / _GLB_N_;
			gg0 = gg1;

			x0 = x1;
			itr ++;
		}
	}
	//END NCG

end:
	// Get timining and metrics
	double t_run = (clock() - t_start) / (double) CLOCKS_PER_SEC;
	double rate = (double)_GLB_N_ / t_run;
	t_lineSearch = t_lineSearch;

	gpu::unalloc(_space, space);
	gpu::unalloc(_space);

	double x_max = *max_element(std::begin(x1), std::end(x1));
	double x_min = *min_element(std::begin(x1), std::end(x1));

	// Output
	json.append("size", _GLB_N_);
	json.append("itr", itr);
	json.append("conv", tol);
	json.append("run_time", t_run);
	json.append("line_search_time", t_lineSearch);
	json.append("rate", rate);
	json.append("x_max", x_max);
	json.append("x_min", x_min);
	
	json.append("space", space);
	json.append("func_val", func_val);

	if (showX) {
		json.append("x", x1);
	}

	cout << "\n\n";
	cout << json.dump();
	cout << "\n\n";
	return 0;
}