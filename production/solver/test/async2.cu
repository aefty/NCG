// Copyright 2012 NVIDIA Corporation

// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at

//     http://www.apache.org/licenses/LICENSE-2.0

// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.


#include <hip/hip_runtime.h>
#include <stdio.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result) {
  #if defined(DEBUG) || defined(_DEBUG)

  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }

  #endif
  return result;
}

__global__ void kernel(double* a, int offset) {
  int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
  double x = (double)i;
  double s = sinf(x);
  double c = cosf(x);
  a[i] = a[i] + sqrtf(s * s + c * c);
}

double maxError(double* a, int n) {
  double maxE = 0;

  for (int i = 0; i < n; i++) {
    double error = fabs(a[i] - 1.0);

    if (error > maxE) { maxE = error; }
  }

  return maxE;
}

int main(int argc, char** argv) {
  const int n = 4 * 1024;
  const int blockSize = 256, nStreams = n;
  const int streamSize = n / nStreams;
  const int streamBytes = streamSize * sizeof(double);
  const int bytes = n * sizeof(double);

  int devId = 0;

  if (argc > 1) { devId = atoi(argv[1]); }

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda( hipSetDevice(devId) );

  // allocate pinned host memory and device memory
  double* a, *d_a;
  checkCuda( hipHostMalloc((void**)&a, bytes, hipHostMallocDefault) );      // host pinned
  checkCuda( hipMalloc((void**)&d_a, bytes) ); // device

  float ms; // elapsed time in milliseconds

  // create events and streams
  hipEvent_t startEvent, stopEvent;
  hipStream_t stream[nStreams];
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );


  // asynchronous version 1: loop over {copy, kernel, copy}
  memset(a, 0, bytes);
  checkCuda( hipEventRecord(startEvent, 0) );

  for (int i = 0; i < nStreams; ++i) {

    checkCuda( hipStreamCreate(&stream[i]) );

    int offset = 0;//i * streamSize;

    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset],
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );
    kernel <<< streamSize / blockSize, blockSize, 0, stream[i] >>> (d_a, offset);
    checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset],
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }

  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));



  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );


  for (int i = 0; i < nStreams; ++i) {
    checkCuda( hipStreamDestroy(stream[i]) );
  }

  hipFree(d_a);
  hipHostFree(a);

  return 0;
}