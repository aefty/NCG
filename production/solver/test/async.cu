// Copyright 2012 NVIDIA Corporation

// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at

//     http://www.apache.org/licenses/LICENSE-2.0

// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.


#include <hip/hip_runtime.h>
#include <stdio.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result) {
  #if defined(DEBUG) || defined(_DEBUG)

  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }

  #endif
  return result;
}

__global__ void kernel(double* a, int offset) {
  int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
  double x = (double)i;
  double s = sinf(x);
  double c = cosf(x);
  a[i] = a[i] + sqrtf(s * s + c * c);
}

double maxError(double* a, int n) {
  double maxE = 0;

  for (int i = 0; i < n; i++) {
    double error = fabs(a[i] - 1.0f);

    if (error > maxE) { maxE = error; }
  }

  return maxE;
}

int main(int argc, char** argv) {
  int M = 1; int F = 1;

  if (argc > 1) {  M = atoi(argv[1]); }

  if (argc > 2) {  F = atoi(argv[2]); }

  const int n = 1024 * M;
  const int blockSize = F, nStreams = n;
  const int streamSize = n / nStreams;
  const int streamBytes = streamSize * sizeof(double);
  const int bytes = n * sizeof(double);

  int devId = 0;

  if (argc > 1) { devId = atoi(argv[1]); }

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  printf("CUDA version : v%f\n", CUDART_VERSION);
  checkCuda( hipSetDevice(devId) );

  // allocate pinned host memory and device memory
  double* a, *d_a;
  checkCuda( hipHostMalloc((void**)&a, bytes, hipHostMallocDefault) );      // host pinned
  checkCuda( hipMalloc((void**)&d_a, bytes) ); // device

  float ms; // elapsed time in milliseconds

  // create events and streams
  hipEvent_t startEvent, stopEvent, dummyEvent;
  hipStream_t stream[nStreams];
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  checkCuda( hipEventCreate(&dummyEvent) );

  for (int i = 0; i < nStreams; ++i) {
    checkCuda( hipStreamCreate(&stream[i]) );
  }

  // baseline case - sequential transfer and execute
  memset(a, 0, bytes);

  checkCuda( hipEventRecord(startEvent, 0) );
  checkCuda( hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice) );
  kernel <<< n / blockSize, blockSize >>> (d_a, 0);

  checkCuda( hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost) );
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for sequential transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // asynchronous version 1: loop over {copy, kernel, copy}
  memset(a, 0, bytes);
  checkCuda( hipEventRecord(startEvent, 0) );

  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset],
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );
    kernel <<< streamSize / blockSize, blockSize, 0, stream[i] >>> (d_a, offset);
    checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset],
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }

  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // asynchronous version 2:
  // loop over copy, loop over kernel, loop over copy
  memset(a, 0, bytes);
  checkCuda( hipEventRecord(startEvent, 0) );

  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset],
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );
  }

  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    kernel <<< streamSize / blockSize, blockSize, 0, stream[i] >>> (d_a, offset);
  }

  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset],
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }

  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V2 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipEventDestroy(dummyEvent) );

  for (int i = 0; i < nStreams; ++i) {
    checkCuda( hipStreamDestroy(stream[i]) );
  }

  hipFree(d_a);
  hipHostFree(a);

  return 0;
}