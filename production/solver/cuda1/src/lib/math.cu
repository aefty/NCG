#include "hip/hip_runtime.h"
namespace cuda {

   //
   __global__ void initSpace( long int N ,  double* x, double EPS, double* space) {
      int i = blockDim.x * blockIdx.x + threadIdx.x;
      int j = blockDim.y * blockIdx.y + threadIdx.y;
      space[j * N + i] = x[i];
   };

   __global__ void bulkGrad( long int N ,  double EPS, double* space, double* grad) {
      int i = blockDim.x * blockIdx.x + threadIdx.x;
      double val = 0.0;

      if (i < N) {
         space[i * N + i] -= EPS;
         FUNCTION(N, &space[i * N], &val);

         space[i * N + i] += 2.0 * EPS;
         val = val * -1.0;

         FUNCTION(N, &space[i * N], &val);
         grad[i] = val / (2.0 * EPS);
      }
   };


   inline void linalg_grad( long int N ,  double EPS, vector<double>& x,  vector<double>& grad, double* _space) {

      int TPB_OPTIMAL_1D = 256;
      int TPB_OPTIMAL_2D = 16;

      double* _grad = (double*)cuda::alloc(grad);
      double* _x = (double*)cuda::alloc(x);

      dim3 GPU_TPB_1D (TPB_OPTIMAL_1D);
      dim3 GPU_BLOCK_1D(_GLB_N_ / GPU_TPB_1D.x);

      dim3 GPU_TPB_2D(TPB_OPTIMAL_2D, TPB_OPTIMAL_2D);
      dim3 GPU_BLOCK_2D(_GLB_N_ / GPU_TPB_2D.x , _GLB_N_ / GPU_TPB_2D.y);

      initSpace <<< GPU_BLOCK_2D , GPU_TPB_2D>>> (N, _x , EPS , _space);
      bulkGrad  <<< GPU_BLOCK_1D , GPU_TPB_1D>>> (N, EPS, _space, _grad);

      cuda::unalloc(_grad, grad);
      cuda::unalloc(_x);
   };

   inline void linalg_grad_spaceAlloc( long int N ,  double EPS, vector<double>& x,  vector<double>& grad, double* _space) {

      int TPB_OPTIMAL_1D = 256;
      int TPB_OPTIMAL_2D = 16;

      double* _grad = (double*)cuda::alloc(grad);
      double* _x = (double*)cuda::alloc(x);

      dim3 GPU_TPB_1D (TPB_OPTIMAL_1D);
      dim3 GPU_BLOCK_1D(_GLB_N_ / GPU_TPB_1D.x);

      dim3 GPU_TPB_2D(TPB_OPTIMAL_2D, TPB_OPTIMAL_2D);
      dim3 GPU_BLOCK_2D(_GLB_N_ / GPU_TPB_2D.x , _GLB_N_ / GPU_TPB_2D.y);

      // initSpace <<< GPU_BLOCK_2D , GPU_TPB_2D>>> (N, _x , EPS , _space);
      bulkGrad  <<< GPU_BLOCK_1D , GPU_TPB_1D>>> (N, EPS, _space, _grad);

      cuda::unalloc(_grad, grad);
      cuda::unalloc(_x);
   };
};

