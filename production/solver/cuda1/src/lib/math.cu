#include "hip/hip_runtime.h"
namespace cuda {

   //
   __global__ void initSpace( long int N ,  double* x, double EPS, double* space) {
      int i = blockDim.x * blockIdx.x + threadIdx.x;
      int j = blockDim.y * blockIdx.y + threadIdx.y;
      space[j * N + i] = x[i] + EPS;
   };

   __global__ void bulkGrad( long int N ,  double EPS, double* space, double* grad) {
      int i = blockDim.x * blockIdx.x + threadIdx.x;
      double val = 0.0;

      if (i < N) {
         space[i * N + i] -= EPS;
         FUNCTION(N, &space[i * N], &val);

         space[i * N + i] += 2.0 * EPS;
         val = val * -1.0;

         FUNCTION(N, &space[i * N], &val);
         grad[i] = val / (2.0 * EPS);
      }
   };


   /*   __global__ void add( long int N ,  double a, double* A, double b, double* B, double* rtrn) {
         int i = blockDim.x * blockIdx.x + threadIdx.x;
         rtrn[i] = a * A[i]  + b * B[i];
      };
   */

   inline void linalg_grad_async(const double& a, vector<double>& A,  const double& b, vector<double>& B, vector<double>& rtrn) {

      int TPB_OPTIMAL_1D = 256;
      int TPB_OPTIMAL_2D = 16;

      dim3 GPU_TPB_1D (TPB_OPTIMAL_1D);
      dim3 GPU_BLOCK_1D(_GLB_N_ / GPU_TPB_1D.x);

      double* _A = (double*)cuda::alloc(A.size() * sizeof(double));
      double* _B = (double*)cuda::alloc(B.size() * sizeof(double));

      add  <<<1 , 1, 0,>>> (N, a, _A, b, _B, rtrn);

      cuda::unalloc(_grad, grad);
      cuda::unalloc(_x);
   };


   inline void linalg_grad( long int N ,  double EPS, vector<double>& x,  vector<double>& grad, double* _space) {

      int TPB_OPTIMAL_1D = 256;
      int TPB_OPTIMAL_2D = 16;

      double* _grad = (double*)cuda::alloc(grad);
      double* _x = (double*)cuda::alloc(x);

      dim3 GPU_TPB_1D (TPB_OPTIMAL_1D);
      dim3 GPU_BLOCK_1D(_GLB_N_ / GPU_TPB_1D.x);

      dim3 GPU_TPB_2D(TPB_OPTIMAL_2D, TPB_OPTIMAL_2D);
      dim3 GPU_BLOCK_2D(_GLB_N_ / GPU_TPB_2D.x , _GLB_N_ / GPU_TPB_2D.y);

      initSpace <<< GPU_BLOCK_2D , GPU_TPB_2D>>> (N, _x , EPS , _space);
      bulkGrad  <<< GPU_BLOCK_1D , GPU_TPB_1D>>> (N, EPS, _space, _grad);

      cuda::unalloc(_grad, grad);
      cuda::unalloc(_x);
   };
};

